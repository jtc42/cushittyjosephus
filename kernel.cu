#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

extern "C" {
	__global__ void gpuMain(int *dest, int n_players, float prob)
	{
		const int id = blockDim.x*blockIdx.x + threadIdx.x;

		/*Set up random number generator*/
		hiprandState state;
		hiprand_init((unsigned long long)clock() + id, 0, 0, &state);

		int plyrs[{{ MAX_PLYRS }}] = { 0 }; /*Creates an array to store player status.
								 Number of players cannot exceed this length.
								 All future references to the number of players 
								 will rely on n_players, NOT the array length.*/

		int survivors = n_players; /*Initial number of survivors*/

		while (survivors > 1) {
			int i = 0;
			for (i = 0; i < n_players; i++) {
				if (plyrs[i] != 1) { /*if player is alive*/
					bool has_killed = false;
					int next_space = 1;
					while (has_killed == false) {
						int next_player_position;
						if (i + next_space >= n_players) {
							next_player_position = i + next_space - n_players;
						}
						else {
							next_player_position = i + next_space;
						}
						if (plyrs[next_player_position] != 1) { /*if next player is alive*/
							float dice_roll = hiprand_uniform_double(&state);
							if (dice_roll <= prob) {
								plyrs[next_player_position] = 1;
								survivors -= 1;
							}
							has_killed = true;
						}
						else {  /*if next player is dead*/
							next_space += 1;
						}
					}
				}
			}
		}

		/*Print out final array*/
		int j;
		int winner;
		for (j = 0; j < n_players; j++) {
			if (plyrs[j] == 0) {
				winner = j;
			}
		}

		dest[id] = winner;
	}
}